#include "common.h"

template<typename T>
device_vector_holder<T>::~device_vector_holder(){
    __free();
}

template<typename T>
void device_vector_holder<T>::__free(){
    if(valid){
        hipFree(__gpu_memory);
        valid = false;
        __size = 0;
    }
}

template<typename T>
device_vector_holder<T>::device_vector_holder(size_t size_, T init)
{
    __malloc(size_);
    thrust::fill(begin_thr(), end_thr(), init);
}

template<typename T>
void device_vector_holder<T>::__malloc(size_t size_){
    if(valid) __free();
    hipMalloc((void**)&__gpu_memory, size_ * sizeof(T));
    __size = size_;
    valid = true;
}

template<typename T>
device_vector_holder<T>::device_vector_holder(size_t size_){
    __malloc(size_);
}

template class device_vector_holder<Vec3f>;

#include "pcd_scene/pcd_scene.h"
template class device_vector_holder<Node_kdtree>;
